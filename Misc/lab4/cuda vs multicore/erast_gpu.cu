#include "hip/hip_runtime.h"
#include<cutil_inline.h>
#include<hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include <sys/types.h>
#include <sched.h>
#include <semaphore.h>
#include <sys/time.h>
#include <time.h>

#define CUDA_GRID_INVALIDATE
//#define THREAD_GRID_INVALIDATE
//#define SEQUENTIAL_INVALIDATE
//#define SEQUENTIAL_GRID_INVALIDATE


#define NUM_THREADS 16  // NOTE THAT THIS MUST BE DIVISIBLE BY 2!
#define NUM_LOCKS (4*NUM_THREADS)

pthread_t threads[NUM_THREADS];
unsigned int threadID[NUM_THREADS];

typedef struct _threadArgs
{
    unsigned long long int thread_idx;
    unsigned long long int j;
    unsigned long long int p;
} threadArgsType;

threadArgsType threadarg[NUM_THREADS];
pthread_attr_t fifo_sched_attr;
pthread_attr_t orig_sched_attr;
struct sched_param fifo_param;

#define MAX (1000000ULL)
//#define MAX (500ULL)
//#define MAX (10000000ULL)
//#define MAX (10000000000ULL)
#define CODE_LENGTH ((sizeof(unsigned char))*8ULL)
#define SCHED_POLICY SCHED_RR

//unsigned char isprime[(MAX/(CODE_LENGTH))+1];
unsigned char *isprime;
sem_t updateIsPrime[NUM_LOCKS];

#ifdef CUDA_GRID_INVALIDATE
unsigned char* isprime_cuda;
unsigned int* threadID_cuda;
unsigned long long int* p_cuda;
__shared__ int iii;
#endif

void print_scheduler(void)
{
    int schedType = sched_getscheduler(getpid());

    switch(schedType)
    {
        case SCHED_FIFO:
            printf("Pthread policy is SCHED_FIFO\n");
            break;
        case SCHED_OTHER:
            printf("Pthread policy is SCHED_OTHER\n");
            break;
        case SCHED_RR:
            printf("Pthread policy is SCHED_RR\n");
            break;
        default:
            printf("Pthread policy is UNKNOWN\n");
    }
}


void set_scheduler(void)
{
    int max_prio, scope, rc;

    print_scheduler();

    pthread_attr_init(&fifo_sched_attr);
    pthread_attr_setinheritsched(&fifo_sched_attr, PTHREAD_EXPLICIT_SCHED);
    pthread_attr_setschedpolicy(&fifo_sched_attr, SCHED_POLICY);

    max_prio=sched_get_priority_min(SCHED_POLICY);
    fifo_param.sched_priority=max_prio;    

    if((rc=sched_setscheduler(getpid(), SCHED_POLICY, &fifo_param)) < 0)
        perror("sched_setscheduler");

    pthread_attr_setschedparam(&fifo_sched_attr, &fifo_param);

    print_scheduler();
}


int chk_isprime(unsigned long long int i)
{
    unsigned long long int idx;
    unsigned int bitpos;

    idx = i/(CODE_LENGTH);
    bitpos = i % (CODE_LENGTH);

    //printf("i=%llu, idx=%llu, bitpos=%u\n", i, idx, bitpos);

    return(((isprime[idx]) & (1<<bitpos))>0);
}

int set_isprime(unsigned long long int i, unsigned char val)
{
    unsigned long long int idx;
    unsigned int bitpos;

    idx = i/(CODE_LENGTH);
    bitpos = i % (CODE_LENGTH);

    //printf("i=%llu, idx=%llu, bitpos=%u\n", i, idx, bitpos);

    // MUTEX PROTECT THIS TEST AND SET
    if(val > 0)
    {
        sem_wait(&updateIsPrime[idx % NUM_LOCKS]);
        isprime[idx] = isprime[idx] | (1<<bitpos);
        sem_post(&updateIsPrime[idx % NUM_LOCKS]);
    }
    else
    {
        sem_wait(&updateIsPrime[idx % NUM_LOCKS]);
        isprime[idx] = isprime[idx] & (~(1<<bitpos));
        sem_post(&updateIsPrime[idx % NUM_LOCKS]);
    }
    // MUTEX PROTECT THIS TEST AND SET
}


void print_isprime(void)
{
    long long int idx=0;

    printf("idx=%lld\n", (MAX/(CODE_LENGTH)));

    for(idx=(MAX/(CODE_LENGTH)); idx >= 0; idx--)
    {
        printf("idx=%lld, %02X\n", idx, isprime[idx]);
    }
    printf("\n");

}


void *invalidate_thread(void *threadptr)
{
    threadArgsType thargs=*((threadArgsType *)threadptr);

    for(thargs.j=2*(thargs.p); (thargs.j < MAX+1); thargs.j+=NUM_THREADS*(thargs.p))
    {
        if((thargs.j+(thargs.thread_idx*thargs.p)) > MAX+1) break;
        set_isprime(thargs.j+((thargs.thread_idx)*(thargs.p)),0);
        //printf("thread %llu, j=%llu\n", thargs.thread_idx, thargs.j);
    }

    //printf("invalidate_thread %llu FINAL j=%llu\n", thargs.thread_idx, thargs.j);
    pthread_exit(&thargs.j);
}

#if defined(CUDA_GRID_INVALIDATE)
__global__ void invalidate_cuda(unsigned long long int* p_cuda,unsigned char* isprime_cuda,unsigned int* threadID_cuda){
	int j,idx;
	unsigned long long int byteidx;
	unsigned int bitpos;	

	idx=blockIdx.x*blockDim.x+threadIdx.x;
	for(j=2*(*p_cuda); j < MAX+1; j+=NUM_THREADS*(*p_cuda)){
        if((j+(idx*(*p_cuda))) > MAX+1)break;        		
		byteidx = 	(j+(idx*(*p_cuda)))	/(CODE_LENGTH);
		bitpos = 	(j+(idx*(*p_cuda)))	%(CODE_LENGTH);
		isprime_cuda[byteidx] = isprime_cuda[byteidx] & (~(1<<bitpos));		
		//atomicAnd((isprime_cuda+byteidx),(~(1<<bitpos)));
    }	
}

__global__ void find_next_lowest_prime_cuda(unsigned long long int* p_cuda, unsigned char* isprime_cuda){
	int j;
	unsigned long long int byteidx;
	unsigned int bitpos;	
	for(j=(*p_cuda)+1; j<MAX+1; j++){		
		byteidx = j/(CODE_LENGTH);
		bitpos = j % (CODE_LENGTH);
		if((((isprime_cuda[byteidx]) & (1<<bitpos))>0)){
			(*p_cuda)=j; break;
		}		
	}
}

#endif


int main(void)
{
		struct timeval tv_start, tv_end;
        unsigned long long int i, j, final_thread_j;
        unsigned long long int p=2;		
        unsigned int cnt=0;
        unsigned long long int thread_idx=0;		

        printf("max uint = %u\n", (0xFFFFFFFF));
        printf("max long long = %llu\n", (0xFFFFFFFFFFFFFFFFULL));

        set_scheduler();
        if(!((isprime=(unsigned char*)malloc((size_t)(MAX/(CODE_LENGTH))+1)) > 0))
        {
            perror("malloc");
            exit(-1);
        }

        int idx=0, ridx=0;

        // Divide the packed isprime binary array into a number of locking
        // regions.
        //
        for(idx=0;idx<NUM_LOCKS;idx++)
        {
            if (sem_init (&updateIsPrime[idx], 0, 0))
	    {
	        perror ("sem_init");
	        printf ("Failed to initialize updateIsPrime semaphore %d\n", idx);
	        exit (-1);
	    }
            sem_post(&updateIsPrime[idx]);
        }

        // Now scramble thread indices into thread IDs that are non-consecutive
        // to reduce lock conention
        for(idx=0, ridx=(NUM_THREADS-1); idx<NUM_THREADS; idx+=2, ridx-=2)
        {
            if(((idx % 2) == 0) || (idx == 0))
            {
                threadID[idx]=ridx;
                threadID[idx+1]=idx;
                //printf("** idx=%d, ridx=%d, threadID[%d]=%d, threadID[%d]=%d\n", idx, ridx, idx, threadID[idx], idx+1, threadID[idx+1]);
            }
            else
            {
                threadID[idx]=ridx;
                threadID[idx+1]=idx;
                //printf("** idx=%d, ridx=%d, threadID[%d]=%d, threadID[%d]=%d\n", idx, ridx, idx, threadID[idx], idx+1, threadID[idx+1]);
            }
        }
 
        for(idx=0, ridx=(NUM_THREADS-1); idx<NUM_THREADS; idx++, ridx--)
            printf("idx=%d, ridx=%d, threadID=%d\n", idx, ridx, threadID[idx]);

        // Not prime by definition
        // 0 & 1 not prime, 2 is prime, 3 is prime, assume others prime to start
        isprime[0]=0xFC; 
        for(i=2; i<MAX; i++) { set_isprime(i, 1); }
  
        //for(i=0; i<MAX; i++) { printf("isprime=%d\n", chk_isprime(i)); }
        //print_isprime();
		
#ifdef CUDA_GRID_INVALIDATE
		int retval;
		dim3 grid(1,1);
		dim3 block(NUM_THREADS,1);
		// Allocate Memory in GPU
		if(hipSuccess!=hipMalloc(&isprime_cuda,(size_t)(MAX/(CODE_LENGTH)+1))){
			perror("CUDA Malloc: isprime_cuda"); exit(-1);
		}
		if(hipSuccess!=hipMalloc(&threadID_cuda,(size_t)(NUM_THREADS))){
			perror("CUDA Malloc: threadID_cuda"); 
			hipFree(isprime_cuda);
			exit(-1);
		}
		if(hipSuccess!=hipMalloc(&p_cuda,(size_t)(sizeof(unsigned long long int)))){
			perror("CUDA Malloc: threadID_cuda");
			hipFree(isprime_cuda);
			hipFree(threadID_cuda);
			exit(-1);
		}
		// Copy from Host to GPU
		if(hipSuccess!=hipMemcpy((void*)isprime_cuda,(const void*)isprime
			,(size_t)((MAX/(CODE_LENGTH))+1),hipMemcpyHostToDevice)){
			perror("CUDAMemCpy: isprime");
			hipFree(isprime_cuda); hipFree(threadID_cuda); hipFree(p_cuda);
			exit(-1);
		}
		if(hipSuccess!=hipMemcpy((void*)threadID_cuda,(const void*)threadID
			,(size_t)(NUM_THREADS),hipMemcpyHostToDevice)){
			perror("CUDAMemCpy: threadID");
			hipFree(isprime_cuda); hipFree(threadID_cuda); hipFree(p_cuda);
			exit(-1);
		}

		if(hipSuccess!=hipMemcpy((void*)p_cuda,(const void*)&p
			,(size_t)(sizeof(unsigned long long int)),hipMemcpyHostToDevice)){
			perror("CUDAMemCpy: p_cuda");
			hipFree(isprime_cuda);
			hipFree(threadID_cuda);
			hipFree(p_cuda);
			exit(-1);
		}

		hipDeviceSynchronize();
#endif
		gettimeofday(&tv_start,NULL);		
        while( (p*p) <=  MAX)
        {
            //printf("p=%llu\n", p);

            // invalidate all multiples of lowest prime so far
            // 
            // simple to compose into a grid of invalidations
            //

#if defined(SEQUENTIAL_INVALIDATE)

            for(j=2*p; j<MAX+1; j+=p)
            {
                //printf("j=%llu\n", j);
                set_isprime(j,0);
            }

#else
#if defined(THREAD_GRID_INVALIDATE)

#warning THREAD_GRID_INVALIDATE

            // Create threads so as to keep them in different regions of
            // the grid to avoid locking as much as possible.
            //
            for(thread_idx=0; thread_idx < NUM_THREADS; thread_idx++)
            {
                    // This is where we permute the thread mapping
                    // to grid mapping to avoid lock contention
                    threadarg[thread_idx].thread_idx=threadID[thread_idx];
                    threadarg[thread_idx].p=p;
                    threadarg[thread_idx].j=2*p;
                    pthread_create(&threads[thread_idx], (const pthread_attr_t *)0, invalidate_thread, (void *)&threadarg[thread_idx]);

            }
    
            for(thread_idx=0; thread_idx < NUM_THREADS; thread_idx++)
            {
                    if(pthread_join(threads[thread_idx], (void **)&final_thread_j) < 0)
                    {
                        perror("pthread_join");
                        exit(-1);
                    }
                    else
                    {
                        //printf("join thread %llu FINAL j=%llu\n", thread_idx, final_thread_j);
                    }
            }

#else
#if defined(SEQUENTIAL_GRID_INVALIDATE)
#warning SEQUENTIAL_GRID_INVALIDATE

            for(thread_idx=0; thread_idx < NUM_THREADS; thread_idx++)
            {
                for(j=2*p; j<MAX+1; j+=NUM_THREADS*p)
                {
                    if((j+(thread_idx*p)) > MAX+1) break;
                    set_isprime(j+(thread_idx*p),0);
                    //printf("thread %llu, j=%llu\n", thread_idx, j);
                }

                //printf("sequential FINAL j=%llu\n", j);
            }

#else
#if defined(CUDA_GRID_INVALIDATE)
					
			invalidate_cuda<<<grid,block>>>(p_cuda,isprime_cuda,threadID_cuda);
			hipDeviceSynchronize();
			find_next_lowest_prime_cuda<<<1,1>>>(p_cuda,isprime_cuda);			
			hipDeviceSynchronize();
//printf("\ncudaErrorInvalidValue=%d\ncudaErrorInvalidDevicePointer=%d\ncudaErrorInvalidMemcpyDirection=%d\nSuccess=%d"
//	, hipErrorInvalidValue,hipErrorInvalidDevicePointer,hipErrorInvalidMemcpyDirection,hipSuccess);
			if(hipSuccess!=hipMemcpy((void*)(&p),(const void*)p_cuda
				,(size_t)(sizeof(unsigned long long int)),hipMemcpyDeviceToHost)){
				perror("CUDAMemCpy: p");
				hipFree(isprime_cuda);
				hipFree(threadID_cuda);
				hipFree(p_cuda);
				exit(-1);
			}
			hipDeviceSynchronize();
			
#endif
#endif
#endif
#endif

#ifndef CUDA_GRID_INVALIDATE
            // find next lowest prime - sequential process
            for(j=p+1; j<MAX+1; j++)
            {
                if(chk_isprime(j)) { p=j; break; }
            }
#endif
//printf("P=%d\n",p);
//break;
        }	// while
		gettimeofday(&tv_end,NULL);
		printf("Time elapsed= %f ms\n",(float)(1000000*tv_end.tv_sec+tv_end.tv_usec-1000000*tv_start.tv_sec-tv_start.tv_usec)/1000.0);
#ifdef CUDA_GRID_INVALIDATE
		if(hipSuccess!=hipMemcpy((void*)isprime,(const void*)isprime_cuda
			,(size_t)((MAX/(CODE_LENGTH))+1),hipMemcpyDeviceToHost)){
			perror("CUDAMemCpy: isprime- Device to host");
			hipFree(isprime_cuda); hipFree(threadID_cuda); hipFree(p_cuda);
			exit(-1);
		}
		hipDeviceSynchronize();
		hipFree(isprime_cuda);
		hipFree(threadID_cuda);
		hipFree(p_cuda);
		hipDeviceReset();		
#endif
printf("\n\n");
        for(i=0; i<MAX+1; i++)
        {
            if(chk_isprime(i))
            { 
                cnt++; 
//                printf("i=%llu\n", i); 
            }
        }		

//printf("\nisprime:%u\t%0x\n",(unsigned char)(*isprime),(unsigned char)(*(isprime+1)));
		
        printf("\nNumber of primes [0..%llu]=%u\n\n", MAX, cnt);
		

        return (i);
}
